#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "padding_partition.hpp"

//
#include <cudf/copying.hpp>
#include <cudf/partitioning.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/scatter.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/utilities/default_stream.hpp>

//
#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

//
#include <cub/device/device_histogram.cuh>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/scatter.h>
#include <thrust/transform.h>


namespace spark_rapids_jni {

struct dispatch_map_type {

  using column_view = cudf::column_view;
  using size_type = cudf::size_type;
  using table = cudf::table;
  using table_view = cudf::table_view;

  template <typename MapType>
  std::enable_if_t<cudf::is_index_type<MapType>(),
                   std::tuple<std::unique_ptr<table>, std::vector<size_type>, std::vector<size_type>>>
  operator()(table_view const& t,
             column_view const& partition_map,
             size_type num_partitions,
             rmm::cuda_stream_view stream,
             rmm::mr::device_memory_resource* mr) const
  {
    // Build a histogram of the number of rows in each partition
    rmm::device_uvector<size_type> histogram(num_partitions + 1, stream);
    std::size_t temp_storage_bytes{};
    std::size_t const num_levels = num_partitions + 1;
    size_type const lower_level  = 0;
    size_type const upper_level  = num_partitions;
    hipcub::DeviceHistogram::HistogramEven(nullptr,
                                        temp_storage_bytes,
                                        partition_map.begin<MapType>(),
                                        histogram.data(),
                                        num_levels,
                                        lower_level,
                                        upper_level,
                                        partition_map.size(),
                                        stream.value());

    rmm::device_buffer temp_storage(temp_storage_bytes, stream);

    hipcub::DeviceHistogram::HistogramEven(temp_storage.data(),
                                        temp_storage_bytes,
                                        partition_map.begin<MapType>(),
                                        histogram.data(),
                                        num_levels,
                                        lower_level,
                                        upper_level,
                                        partition_map.size(),
                                        stream.value());

    // do padding only if there exists nullable columns
    bool padding_or_not = std::any_of(t.begin(), t.end(), [](const cudf::column_view &cv)
                                      { return cv.nullable(); });
    size_type padding_rows = 0;

    if (padding_or_not) {
      // do padding only if there exists partition offset which % 8 != 0
      auto padding_delta = [] __host__ __device__(size_type v) {
        return ((v + 7) >> 3 << 3) - v;
      };
      padding_rows = thrust::reduce(
        rmm::exec_policy(stream), 
        thrust::make_transform_iterator(histogram.begin(), padding_delta),
        thrust::make_transform_iterator(histogram.end(), padding_delta)
      );
      padding_or_not = padding_rows > 0;
    }

    // building the offsets of each partition through padding and accumulation
    rmm::device_uvector<size_type> offsets(num_partitions + 1, stream);

    if (padding_or_not) {
      // padding the partitions' intervals with 8 before making offsets from them  
      thrust::transform(rmm::exec_policy(stream),
        histogram.begin(), histogram.end(), offsets.begin(),
        [] __device__(auto v) { return (v + 7) >> 3 << 3; });
    } else {
      thrust::transform(rmm::exec_policy(stream),
        histogram.begin(), histogram.end(), offsets.begin(),
        thrust::identity<size_type>());
    }

    // `histogram` was created with an extra entry at the end such that an
    // exclusive scan will put the total number of rows at the end
    thrust::exclusive_scan(
      rmm::exec_policy(stream), offsets.begin(), offsets.end(), offsets.begin());

    // Copy offsets to host before the transform below modifies it
    auto const partition_offsets = cudf::detail::make_std_vector_sync(offsets, stream);
    // Copy lengths to host before the transform below modifies it
    auto const partition_lengths = cudf::detail::make_std_vector_sync(
        cudf::device_span<size_type const>{histogram.data(), histogram.size() - 1},
        stream);

    // Unfortunately need to materialize the scatter map because
    // `detail::scatter` requires multiple passes through the iterator
    rmm::device_uvector<size_type> scatter_map(partition_map.size(), stream);

    // For each `partition_map[i]`, atomically increment the corresponding
    // partition offset to determine `i`s location in the output
    thrust::transform(rmm::exec_policy(stream),
                      partition_map.begin<MapType>(),
                      partition_map.end<MapType>(),
                      scatter_map.begin(),
                      [offset_data = offsets.data()] __device__(auto partition_number) {
                        return atomicAdd(&offset_data[partition_number], 1);
                      });

    // Apply scattering if padding is not necessary
    if (not padding_or_not) {
      auto scattered = cudf::detail::scatter(t, scatter_map, t, stream, mr);

      return {std::move(scattered), std::move(partition_offsets), std::move(partition_lengths)};
    }

    // Convert scattering to gathering for padding
    auto original_size = scatter_map.size();
    auto padded_size = scatter_map.size() + padding_rows;

    // initialize gather_map with index 0
    rmm::device_uvector<size_type> gather_map(padded_size, stream);
    thrust::uninitialized_fill(rmm::exec_policy_nosync(stream),
                               gather_map.begin(),
                               gather_map.end(),
                               0);

    // Convert scatter map to a gather map
    thrust::scatter(rmm::exec_policy_nosync(stream),
                    thrust::make_counting_iterator<size_type>(0),
                    thrust::make_counting_iterator<size_type>(original_size),
                    scatter_map.begin(),
                    gather_map.begin());

    auto gathered = cudf::detail::gather(t,
                                         cudf::device_span<size_type const>(gather_map),
                                         cudf::out_of_bounds_policy::DONT_CHECK,
                                         cudf::detail::negative_index_policy::NOT_ALLOWED,
                                         stream,
                                         mr);

    return {std::move(gathered), std::move(partition_offsets), std::move(partition_lengths)};
  }

  template <typename MapType, typename... Args>
  std::enable_if_t<not cudf::is_index_type<MapType>(),
                   std::tuple<std::unique_ptr<table>, std::vector<size_type>, std::vector<size_type>>>
  operator()(Args&&...) const
  {
    CUDF_FAIL("Unexpected, non-integral partition map.");
  }
};

std::tuple<std::unique_ptr<cudf::table>, std::vector<cudf::size_type>, std::vector<cudf::size_type>> padding_partition(
  cudf::table_view const& t,
  cudf::column_view const& partition_map,
  cudf::size_type num_partitions,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(t.num_rows() == partition_map.size(),
               "Size mismatch between table and partition map.");
  CUDF_EXPECTS(not partition_map.has_nulls(), "Unexpected null values in partition_map.");

  if (num_partitions == 0 or t.num_rows() == 0) {
    // The output offsets vector must have size `num_partitions + 1` as per documentation.
    return {empty_like(t), std::vector<cudf::size_type>(num_partitions + 1, 0), std::vector<cudf::size_type>(num_partitions, 0)};
  }

  return cudf::type_dispatcher(
    partition_map.type(), dispatch_map_type{}, t, partition_map, num_partitions,
    cudf::get_default_stream(), mr);
}

}
